/*
 * MD5 crypt massive dictionary attack (md5crypt-mda)
 * CUDA implementation based on Benjamin Vernoux's MD5 cracker
 * Derived from the RSA Data Security, Inc. MD5 Message Digest Algorithm
 *
 * Yao Wei, mwei@lxde.org
 *
 * ====
 * optimizations planned:
 * - memory coalscing
 * - sorted password list. same password length in single warp.
 *   (which has exactly same iterations without branch diversion)
 * - bank deconflicting
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>

#define HASH_LENGTH 16 /* hash size. may do with bank deconflicting. */
#define SALT_MAX_LENGTH 8
#define PASSWORD_MAX_LENGTH 16
#define DICTIONARY_MAX_SIZE 100000000
#define SHADOW_MAX_SIZE 20000
#define GPU_COUNT 1

#define BLOCK_SIZE 512

/* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define FF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }

#define S11 7
#define S12 12
#define S13 17
#define S14 22
#define S21 5
#define S22 9
#define S23 14
#define S24 20
#define S31 4
#define S32 11
#define S33 16
#define S34 23
#define S41 6
#define S42 10
#define S43 15
#define S44 21

#define x0 0x67452301
#define y0 0xEFCDAB89
#define z0 0x98BADCFE
#define w0 0x10325476

struct md5_ctx {
  unsigned int input[16];
  unsigned int inputSize;
  uint4 hash;
};

__device__ void md5_init(struct md5_ctx *ctx) {
  ctx->inputSize = 0;
  ctx->hash.x = x0;
  ctx->hash.y = y0;
  ctx->hash.z = z0;
  ctx->hash.w = w0;
}

/* md5 update script. must filled up with 128 bytes. */
inline __device__ void md5_calc(struct md5_ctx *ctx) {
  uint4 nhash;

  nhash.x = ctx->hash.x;
  nhash.y = ctx->hash.y;
  nhash.z = ctx->hash.z;
  nhash.w = ctx->hash.w;

	/* Round 1 */
	FF ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[0],  S11, 3614090360); /* 1 */
  FF ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[1],  S12, 3905402710); /* 2 */
  FF ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[2],  S13,  606105819); /* 3 */
  FF ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[3],  S14, 3250441966); /* 4 */
  FF ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[4],  S11, 4118548399); /* 5 */
  FF ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[5],  S12, 1200080426); /* 6 */
  FF ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[6],  S13, 2821735955); /* 7 */
  FF ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[7],  S14, 4249261313); /* 8 */
  FF ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[8],  S11, 1770035416); /* 9 */
  FF ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[9],  S12, 2336552879); /* 10 */
  FF ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[10], S13, 4294925233); /* 11 */
  FF ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[11], S14, 2304563134); /* 12 */
  FF ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[12], S11, 1804603682); /* 13 */
  FF ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[13], S12, 4254626195); /* 14 */
  FF ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[14], S13, 2792965006); /* 15 */
  FF ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[15], S14, 1236535329); /* 16 */

  /* Round 2 */
  GG ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[1],  S21, 4129170786); /* 17 */
  GG ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[6],  S22, 3225465664); /* 18 */
  GG ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[11], S23,  643717713); /* 19 */
  GG ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[0],  S24, 3921069994); /* 20 */
  GG ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[5],  S21, 3593408605); /* 21 */
  GG ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[10], S22,   38016083); /* 22 */
  GG ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[15], S23, 3634488961); /* 23 */
  GG ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[4],  S24, 3889429448); /* 24 */
  GG ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[9],  S21,  568446438); /* 25 */
  GG ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[14], S22, 3275163606); /* 26 */
  GG ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[3],  S23, 4107603335); /* 27 */
  GG ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[8],  S24, 1163531501); /* 28 */
  GG ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[13], S21, 2850285829); /* 29 */
  GG ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[2],  S22, 4243563512); /* 30 */
  GG ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[7],  S23, 1735328473); /* 31 */
  GG ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[12], S24, 2368359562); /* 32 */

  /* Round 3 */
  HH ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[5],  S31, 4294588738); /* 33 */
  HH ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[8],  S32, 2272392833); /* 34 */
  HH ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[11], S33, 1839030562); /* 35 */
  HH ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[14], S34, 4259657740); /* 36 */
  HH ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[1],  S31, 2763975236); /* 37 */
  HH ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[4],  S32, 1272893353); /* 38 */
  HH ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[7],  S33, 4139469664); /* 39 */
  HH ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[10], S34, 3200236656); /* 40 */
  HH ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[13], S31,  681279174); /* 41 */
  HH ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[0],  S32, 3936430074); /* 42 */
  HH ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[3],  S33, 3572445317); /* 43 */
  HH ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[6],  S34,   76029189); /* 44 */
  HH ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[9],  S31, 3654602809); /* 45 */
  HH ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[12], S32, 3873151461); /* 46 */
  HH ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[15], S33,  530742520); /* 47 */
  HH ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[2],  S34, 3299628645); /* 48 */

  /* Round 4 */
  II ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[0],  S41, 4096336452); /* 49 */
  II ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[7],  S42, 1126891415); /* 50 */
  II ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[14], S43, 2878612391); /* 51 */
  II ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[5],  S44, 4237533241); /* 52 */
  II ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[12], S41, 1700485571); /* 53 */
  II ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[3],  S42, 2399980690); /* 54 */
  II ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[10], S43, 4293915773); /* 55 */
  II ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[1],  S44, 2240044497); /* 56 */
  II ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[8],  S41, 1873313359); /* 57 */
  II ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[15], S42, 4264355552); /* 58 */
  II ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[6],  S43, 2734768916); /* 59 */
  II ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[13], S44, 1309151649); /* 60 */
  II ( nhash.x, nhash.y, nhash.z, nhash.w, ctx->input[4],  S41, 4149444226); /* 61 */
  II ( nhash.w, nhash.x, nhash.y, nhash.z, ctx->input[11], S42, 3174756917); /* 62 */
  II ( nhash.z, nhash.w, nhash.x, nhash.y, ctx->input[2],  S43,  718787259); /* 63 */
  II ( nhash.y, nhash.z, nhash.w, nhash.x, ctx->input[9],  S44, 3951481745); /* 64 */

	ctx->hash.x += nhash.x;
	ctx->hash.y += nhash.y;
	ctx->hash.z += nhash.z;
	ctx->hash.w += nhash.w;
}

__device__ void md5_update(struct md5_ctx *ctx, const char *in, int size) {
  char * inputChar = (char *) ctx->input;

  for(int i=0; i<size; i++){
    inputChar[ctx->inputSize++] = in[i];
    if (ctx->inputSize % 64 == 0) md5_calc(ctx);
  }
}

__device__ void md5_final(char * final, struct md5_ctx *ctx){
  char * inputChar = (char *) ctx->input;
  unsigned int realSize = ctx->inputSize;

  inputChar[ctx->inputSize++] = 0x80;
  if (ctx->inputSize % 64 == 0) md5_calc(ctx);

  while(ctx->inputSize % 64 != 56){
    inputChar[ctx->inputSize++] = 0x00;
    if (ctx->inputSize % 64 == 0) md5_calc(ctx);
  }

  ctx->input[14] = realSize << 3;
  ctx->input[15] = realSize >> 29;

  md5_calc(ctx);
  memcpy(final, &(ctx->hash), 16);
}

__device__ unsigned char itoa64[] = /* 0 ... 63 => ascii - 64 */
  "./0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";

__device__ bool to64(char *s, u_int32_t v, int n) {
  while (--n >= 0) {
    if (*s != itoa64[v&0x3f]){
      return false;
    }
    v >>= 6;
    s++;
  }
  return true;
}


__device__ bool md5crypt(const char *pw, const char *salt, const unsigned char saltlength, const char *target) {
	struct md5_ctx ctx;
	char final[16];
  unsigned int pwlen;

  for(pwlen = 0; pwlen < 16 && pw[pwlen] != '\0'; pwlen++);

  /* get first "final" from password+salt+password */
	md5_init(&ctx);
	md5_update(&ctx, pw, pwlen);
	md5_update(&ctx, salt, saltlength);
	md5_update(&ctx, pw, pwlen);
	md5_final(final, &ctx);

  /* get second "final" from password+"$1$"+salt+fractions_of_final+weird_things */
	md5_init(&ctx);
	md5_update(&ctx, pw, pwlen);
	md5_update(&ctx, "$1$", 3);
	md5_update(&ctx, salt, saltlength);

	for (char pl = pwlen; pl > 0; pl -= 16) {
		md5_update(&ctx, final, pl>16 ? 16 : pl);
  }

  memset(final, 0, 16);

	for (char i = pwlen; i != 0; i >>= 1) {
		if(i&1) {
		  md5_update(&ctx, final, 1);
		} else {
		  md5_update(&ctx, pw, 1);
    }
  }

	md5_final(final, &ctx);

	/* 1000 iterations that runs slow in _Pentium 60 MHz_ >w< */
	for(unsigned short i=0;i<1000;i++) {
		md5_init(&ctx);

		if(i & 1) {
      md5_update(&ctx, pw, pwlen);
    } else {
      md5_update(&ctx, final, 16);
    }

		if(i % 3) {
      md5_update(&ctx, salt, saltlength);
    }

		if(i % 7) {
      md5_update(&ctx, pw, pwlen);
    }

		if(i & 1) {
			md5_update(&ctx, final, 16);
    } else {
			md5_update(&ctx, pw, pwlen);
    }

		md5_final(final, &ctx);
	}

  #pragma unroll
  for(int i=0; i<16; i++){
    if (final[i] != target[i]) return false;
  }
  return true;
}

__global__ void md5_crypt_dictionary_attack(const char *t, const char *salt, const unsigned char saltlength, const char *dictionary, const unsigned int * dictionary_index, unsigned int dictionary_index_size, unsigned int *result) {
  if (*result != UINT_MAX) return;
  /* get dictionary word from global memory */
  unsigned int dictionary_id = blockDim.x * blockIdx.x + threadIdx.x;
  __shared__ char target[17];
  if (threadIdx.x < 16)
    target[threadIdx.x] = t[threadIdx.x];

  /* size check */
  if (dictionary_id >= dictionary_index_size){
    return;
  }

  const char *password = &(dictionary[dictionary_index[dictionary_id]]);

  /* attack and examine result */
  bool a = md5crypt(password, salt, saltlength, target);
  if (a == true){
    *result = dictionary_id;
  }
}

unsigned char a64toi[128];
void from64_prepare(){
  int c = 0;
  a64toi['.'] = c++;
  a64toi['/'] = c++;
  for (int i='0'; i<='9'; i++){
    a64toi[i] = c++;
  }
  for (int i='A'; i<='Z'; i++){
    a64toi[i] = c++;
  }
  for (int i='a'; i<='z'; i++){
    a64toi[i] = c++;
  }
}
void from64(char* from, char* to){
  /* 12, 6, 0, 13, 7, 1, 14, 8, 2, 15, 9, 3, 5, 10, 4, 11 */
  int x;
  x = a64toi[from[3]] << 18 | a64toi[from[2]] << 12 | a64toi[from[1]] << 6 | a64toi[from[0]];
  to[12] = x & 0x000000FF; x >>= 8;
  to[6] = x & 0x000000FF; x >>= 8;
  to[0] = x & 0x000000FF; x >>= 8;
  x = a64toi[from[7]] << 18 | a64toi[from[6]] << 12 | a64toi[from[5]] << 6 | a64toi[from[4]];
  to[13] = x & 0x000000FF; x >>= 8;
  to[7] = x & 0x000000FF; x >>= 8;
  to[1] = x & 0x000000FF; x >>= 8;
  x = a64toi[from[11]] << 18 | a64toi[from[10]] << 12 | a64toi[from[9]] << 6 | a64toi[from[8]];
  to[14] = x & 0x000000FF; x >>= 8;
  to[8] = x & 0x000000FF; x >>= 8;
  to[2] = x & 0x000000FF; x >>= 8;
  x = a64toi[from[15]] << 18 | a64toi[from[14]] << 12 | a64toi[from[13]] << 6 | a64toi[from[12]];
  to[15] = x & 0x000000FF; x >>= 8;
  to[9] = x & 0x000000FF; x >>= 8;
  to[3] = x & 0x000000FF; x >>= 8;
  x = a64toi[from[19]] << 18 | a64toi[from[18]] << 12 | a64toi[from[17]] << 6 | a64toi[from[16]];
  to[5] = x & 0x000000FF; x >>= 8;
  to[10] = x & 0x000000FF; x >>= 8;
  to[4] = x & 0x000000FF; x >>= 8;
  x = a64toi[from[21]] << 6 | a64toi[from[20]];
  to[11] = x & 0x000000FF; x >>= 8;
}

void read_file(FILE * file, char** storage, unsigned int* storage_size, unsigned int** index, unsigned int* index_size){
  fseek(file, 0L, SEEK_END);
  *storage_size = ftell(file);
  rewind(file);

  *storage = (char *) malloc(*storage_size+1);
  fread(*storage, *storage_size, 1, file);

  *index_size = 0;
  for(int i=0; i<=*storage_size; i++){
    if ((*storage)[i] == '\n'){
      (*storage)[i] = '\0';
      *index_size += 1;
    }
  }

  *index = (unsigned int *) malloc(sizeof(unsigned int)*(*index_size));

  unsigned int state = 0;
  unsigned int index_number = 0;
  for(unsigned int i=0; i<=*storage_size; i++){
    if ((*storage)[i] != '\0' && state == 0){
      state = 1;
      (*index)[index_number] = i;
      index_number++;
    } else if ((*storage)[i] == '\0' && state == 1){
      state = 0;
    }
  }
}

int main(int argc, char** argv){
  omp_set_num_threads(GPU_COUNT);

  if (argc < 2 || strcmp(argv[1], "-h") == 0){
    printf("%s [dictionary] [shadow]\n", argv[0]);
  }

  FILE *dictionary = fopen(argv[1], "r");
  FILE *shadow = fopen(argv[2], "r");

  if (!dictionary || !shadow){
    fprintf(stderr, "Error opening file. Check if the file correct or not.");
    return -1;
  }

  char* dictionary_mem;
  unsigned int dictionary_size;
  unsigned int* dictionary_index;
  unsigned int dictionary_index_size;
  read_file(dictionary, &dictionary_mem, &dictionary_size, &dictionary_index, &dictionary_index_size);
  printf("dictionary_index_size: %d\n", dictionary_index_size);

  char hash[23];
  char salt[9];
  char hashArray[SHADOW_MAX_SIZE][23];
  char tArray[SHADOW_MAX_SIZE][16];
  char saltArray[SHADOW_MAX_SIZE][9];
  char line[100];
  unsigned int shadow_count = 0;

  from64_prepare();
  while(fgets(line, 100, shadow)){
    if (sscanf(line, "$1$%[^$]$%[^\n]", &salt, &hash) <= 0){
      continue;
    }
    memcpy(hashArray[shadow_count], hash, sizeof(char) * 23);
    from64(hash, tArray[shadow_count]);
    memcpy(saltArray[shadow_count], salt, sizeof(char) * 9);
    shadow_count += 1;
    if (shadow_count >= SHADOW_MAX_SIZE) break;
  }
  fclose(shadow);
  fclose(dictionary);

  unsigned int grid_size = dictionary_size / BLOCK_SIZE;
  if (dictionary_size % BLOCK_SIZE != 0) grid_size += 1;

  #pragma omp parallel
  {
    int gid = omp_get_thread_num();
    hipSetDevice(gid);
    hipFuncSetCacheConfig( reinterpret_cast<const void*>(md5_crypt_dictionary_attack), hipFuncCachePreferL1 );

    char *d_dictionary;
    unsigned int* d_dictionary_index;
    hipMalloc((void**)&d_dictionary, sizeof(char)*dictionary_size);
    hipMalloc((void**)&d_dictionary_index, sizeof(unsigned int)*dictionary_index_size);
    hipMemcpy(d_dictionary, dictionary_mem, sizeof(char)*dictionary_size, hipMemcpyHostToDevice);
    hipMemcpy(d_dictionary_index, dictionary_index, sizeof(unsigned int)*dictionary_index_size, hipMemcpyHostToDevice);

    char *d_t, *d_salt;
    hipMalloc((void**)&d_t, sizeof(char)*16);
    hipMalloc((void**)&d_salt, sizeof(char)*9);

    unsigned int *result;
    hipHostAlloc((void**) &result, sizeof(int), hipHostMallocDefault);

    #pragma omp for
    for(int i=0; i<shadow_count; i++){
      *result = UINT_MAX;

      hipMemcpy(d_t, tArray[i], sizeof(char)*16, hipMemcpyHostToDevice);
      hipMemcpy(d_salt, saltArray[i], sizeof(char)*9, hipMemcpyHostToDevice);

      md5_crypt_dictionary_attack<<<grid_size,BLOCK_SIZE>>>(d_t, d_salt, strlen(saltArray[i]), d_dictionary, d_dictionary_index, dictionary_index_size, result);
      hipDeviceSynchronize();

      printf("[%d]", gid);
      if (*result != UINT_MAX){
        printf("$1$%s$%s = %s\n", saltArray[i], hashArray[i], dictionary_mem+dictionary_index[*result]);
      } else {
        printf("$1$%s$%s = not found\n", saltArray[i], hashArray[i]);
      }
    }

    hipHostFree(result);
    hipFree(d_t);
    hipFree(d_salt);
    hipFree(d_dictionary);
    hipFree(d_dictionary_index);
  }

  free(dictionary_mem);
  free(dictionary_index);
}
